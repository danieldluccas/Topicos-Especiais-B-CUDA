#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

void main(void)
{
	hipDeviceProp_t minhaGPU;
	hipGetDeviceProperties(&minhaGPU, 0);
	//Especificações da minha GPU
	////GEFORCE GTX 1650 (MICROARQUITETURA TURING)
	printf("Nome: %s \n", minhaGPU.name);
	printf("Clock: %.2lf GHz\n", (float) minhaGPU.clockRate/1000000);
	printf("Numero maximo de blocos: %d \n", minhaGPU.maxBlocksPerMultiProcessor);
	printf("Numero max de cada thread por bloco: %d\n", minhaGPU.maxThreadsPerBlock);
	printf("No qtd de multiprocessamento: %d\n", minhaGPU.multiProcessorCount);
	printf("Memoria compartilhada por SM: %.3fKB\n", (float)minhaGPU.sharedMemPerMultiprocessor/1000);
	printf("Memoria global: %.3llf GB\n", (float)minhaGPU.totalGlobalMem / 1000000000);
	printf("Tamanho de warp em threads: %d\n", minhaGPU.warpSize);
	//N cores por SM
	
	//referência: https://docs.nvidia.com/cuda/cuda-runtime-api/structcudaDeviceProp.html
}
