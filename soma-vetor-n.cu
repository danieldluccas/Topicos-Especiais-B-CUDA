#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#define TAMANHO 10
#define BLOCOS 1
#define THREADS 1


__global__ void addVetor(int *c, int *a, int *b)
{
    int i = threadIdx.x;
    c[i] = a[i]+b[i];
    

 /*   printf("teste addkernel %i\n", blockIdx.x * blockDim.x + threadIdx.x);
    int i = threadIdx.x;
    int teste = blockIdx.x * blockDim.x + threadIdx.x;
    printf("i = %i\n", i);
    printf("teste m[%i] = %i\n", i, a[teste]);
  */

}



int main()
{
 
 int m[TAMANHO];
 int n[TAMANHO];
 int resultado[TAMANHO];
 const int teste[TAMANHO] = { 1,2,3,4,5 };
 int testando = 3;
 int* d_a = nullptr;
 int* d_b = nullptr;
 int* d_c = nullptr;

 int* d_t = nullptr;
 
 
 
//atribuição de valores nos vetores
     for (int i = 0; i < TAMANHO; i++)
     {
         m[i] = i;
         n[i] = 2*i;   
     }
     
     printf("====RESERVA A MEMORIA DA GPU PARA DADOS====\n");
//reserva espaço na memoria do device
     hipMalloc((void**)&d_a, TAMANHO * sizeof(int));
     hipMalloc((void**)&d_b, TAMANHO * sizeof(int));
     hipMalloc((void**)&d_c, TAMANHO * sizeof(int));
     //hipMalloc((void**)&d_t, sizeof(int));

     printf("====ENVIO DOS DADOS NA CPU PARA GPU====\n");
//envia os dados da CPU para a memória reservada no device
     hipMemcpy(d_a, m, TAMANHO * sizeof(int), hipMemcpyHostToDevice);
     hipMemcpy(d_b, n, TAMANHO * sizeof(int), hipMemcpyHostToDevice);
     //hipMemcpy(d_t, &testando, sizeof(int), hipMemcpyHostToDevice);
     
     
     
     addVetor << <1, TAMANHO >> > (d_c, d_a, d_b);
     hipDeviceSynchronize();
//retorna os dados do device para a CPU
     hipMemcpy(resultado, d_c, TAMANHO * sizeof(int), hipMemcpyDeviceToHost);

     for (int i = 0; i < TAMANHO; i++)
     {
         printf("c[%d] = %d\n",i, resultado[i]);
     }

     
     hipFree(d_a);
     hipFree(d_b);
     hipFree(d_c);
     
     hipDeviceReset();
     printf("====FIM====\n");
}
