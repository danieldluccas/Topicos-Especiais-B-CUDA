#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

#define TAMANHO 3
#define BLOCOS 3
#define THREADS 1


__global__ void addVetor(int* c, int* a, int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];


    /*   printf("teste addkernel %i\n", blockIdx.x * blockDim.x + threadIdx.x);
       int i = threadIdx.x;
       int teste = blockIdx.x * blockDim.x + threadIdx.x;
       printf("i = %i\n", i);
       printf("teste m[%i] = %i\n", i, a[teste]);
     */

}
__global__ void transposta(int *a, int *r)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x; // blockDim (3) * blockId (0..2) + threadId (0..2)
    //int j = blockIdx.y;
    int k = (blockDim.x * blockIdx.y) + threadIdx.y;
    printf(" y = %d\n", blockIdx.x + threadIdx.y);
    printf("a[%i] = %i \n", i, a[i]);
    printf("b[%i] = %i \n", j, a[j]);
    if (i < TAMANHO*TAMANHO && k < TAMANHO*TAMANHO)
    {    
        int indexa = i + TAMANHO * k;
        int indexb = k + TAMANHO * i;
        r[indexb] = a[indexa];
    }
    
}


int main()
{

    int m[TAMANHO][TAMANHO];
    int resultado[TAMANHO][TAMANHO];
    
    int* d_a = nullptr;
    int* d_r = nullptr;
  

    //atribuição de valores nos vetores
    printf("===MATRIZ GERADA===\n");
    for (int i = 0; i < TAMANHO; i++)
    {
        for (int j = 0; j < TAMANHO; j++)
        {
            m[i][j] = rand() % 100;
            printf("%d ", m[i][j]);
            
        }
        printf("\n");
    }

    printf("====RESERVA A MEMORIA DA GPU PARA DADOS====\n");
    //reserva espaço na memoria do device
    hipMalloc((void**)&d_a, TAMANHO * TAMANHO * sizeof(int));
    hipMalloc((void**)&d_r, TAMANHO * TAMANHO * sizeof(int));

    printf("====ENVIO DOS DADOS NA CPU PARA GPU====\n");
    //envia os dados da CPU para a memória reservada no device
    hipMemcpy(d_a, m, TAMANHO * TAMANHO * sizeof(int), hipMemcpyHostToDevice);


    transposta << <BLOCOS, TAMANHO >> > (d_a, d_r);
    hipDeviceSynchronize();
    //retorna os dados do device para a CPU
    hipMemcpy(resultado, d_r, TAMANHO * TAMANHO * sizeof(int), hipMemcpyDeviceToHost);
    printf("=========\n");
    for (int i = 0; i < TAMANHO*TAMANHO; i++)
    {
        printf("c[%d] = %d\n", i, resultado[i]);
    }


    hipFree(d_a);
    hipFree(d_r);
  
    hipDeviceReset();
    printf("====FIM====\n");
}
