
#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>

#define TAMANHO 5


__global__ void identidade2d(int *a)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    printf("i = %d\n", i);
    printf("j = %d\n", j);
    if (i < TAMANHO && j < TAMANHO)
    {
        if (i == j)
        {
            *(a + (i + j)) = 1;
        }
        else
        {
            *(a+(i + j)) = 0;
        }
        printf("%i\n", *(a + (i + j)));
    }
    
}

int main()
{
    int m[TAMANHO][TAMANHO];
    int *matriz;
    const int n_threads = 1;
    const int blocos = TAMANHO / n_threads;
        
    dim3 grade(1,1);

    /*for (int i = 0; i < TAMANHO; i++)
    {
        for (int j = 0; j < TAMANHO; j++)
        {
            m[i][j] = rand() % 100;
            printf("%4i ", m[i][j]);
        }
        printf("\n");
    }
    */

    //reserva espaço na memória
    hipMalloc((void **)&matriz, TAMANHO * TAMANHO * sizeof(int));


    identidade2d <<<1,1>>> (matriz);
    hipDeviceSynchronize();
    //cudaMemcpy(m, matriz, TAMANHO * TAMANHO * sizeof(int), cudaMemcpyDeviceToHost);
    hipDeviceReset();
    hipFree(matriz);
}

