#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

//threads = tamanho * tamanho
#define TAMANHO 8
#define BLOCOS 1
#define THREADS 25


__global__ void identi (int* b)
{   
    int i = threadIdx.x;
    
    //printf("i: %d\n", i);
    if (i == 0)
        b[i] = 1;
    else if (i % (TAMANHO + 1) == 0)
    {
        b[i] = 1;
    }
    else
    {
        b[i] = 0;
    }
}



int main()
{
   const int basexaltura = TAMANHO * TAMANHO;
   int resultado[basexaltura];
    
    int* d_c = nullptr;


    
    //reserva espaço na memoria do device
    printf("====RESERVA A MEMORIA DA GPU PARA DADOS====\n");
    hipMalloc((void**)&d_c, basexaltura * sizeof(int));
   
    //invoca o kernel
    identi << <BLOCOS, basexaltura >> > (d_c);
    hipDeviceSynchronize();

    //retorna os dados do device para a CPU
    hipMemcpy(resultado, d_c, basexaltura * sizeof(int), hipMemcpyDeviceToHost);

    //imprime a matriz
    for (int i = 0; i < TAMANHO; i++)
    {
        for (int j = 0; j < TAMANHO; j++)
        {
            //printf("c[%d] = %d\n", i * TAMANHO + j + 1, resultado[i*TAMANHO + j]);
            printf("%d ", resultado[i * TAMANHO + j]);
        }
        printf("\n");
    }

    hipFree(d_c);
    hipDeviceReset();
    printf("====FIM====\n");
}