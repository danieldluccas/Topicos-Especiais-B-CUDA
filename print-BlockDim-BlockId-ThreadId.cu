#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


__global__ void callID(void)
{
    printf("Block Dim: %d | Block ID: %d | Thread ID: %d\n", blockDim.x, blockIdx.x, threadIdx.x);
}

int main()
{
    callID << <3, 5 >> > ();
    hipDeviceSynchronize();
    hipDeviceReset();
    return 0;
}
